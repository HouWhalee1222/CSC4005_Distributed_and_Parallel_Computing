#include "hip/hip_runtime.h"
#include <graphic/graphic.hpp>
#include <imgui_impl_sdl.h>
#include <cstring>
#include <chrono>
#include <random>
#include <utility>
#include <iostream>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#define ITERATION 100
#define BODIES 200
#define COLLISION_RATIO 0.01

__device__ __managed__ double x[BODIES];
__device__ __managed__ double y[BODIES];
__device__ __managed__ double vx[BODIES];
__device__ __managed__ double vy[BODIES];
__device__ __managed__ double ax[BODIES];
__device__ __managed__ double ay[BODIES];
__device__ __managed__ double m[BODIES];

__device__ void handle_wall_collision(double position_range, double radius, int index) {
    bool flag = false;
    if (x[index] <= radius) {
        flag = true;
        x[index] = radius + radius * COLLISION_RATIO;
        vx[index] = -vx[index];
    } else if (x[index] >= position_range - radius) {
        flag = true;
        x[index] = position_range - radius - radius * COLLISION_RATIO;
        vx[index] = -vx[index];
    }

    if (y[index] <= radius) {
        flag = true;
        y[index] = radius + radius * COLLISION_RATIO;
        vy[index] = -vy[index];
    } else if (y[index] >= position_range - radius) {
        flag = true;
        y[index] = position_range - radius - radius * COLLISION_RATIO;
        vy[index] = -vy[index];
    }
    if (flag) {
        ax[index] = 0;
        ay[index] = 0;
    }
}

__device__  void update_for_tick(double elapse, double position_range, double radius, int index) {
    
    vx[index] += ax[index] * elapse;
    vy[index] += ay[index] * elapse;
    handle_wall_collision(position_range, radius, index);
    x[index] += vx[index] * elapse;
    y[index] += vy[index] * elapse;
    handle_wall_collision(position_range, radius, index);
}

__device__ void check_and_update(int i, int j, double radius, double gravity) {
    double delta_x = x[i] - x[j];
    double delta_y = y[i] - y[j];
    double distance_square = delta_x * delta_x + delta_y * delta_y;
    double ratio = 1 + COLLISION_RATIO;

    if (distance_square < radius * radius) {
        distance_square = radius * radius;
    }

    auto distance = std::sqrt(distance_square);

    if (distance < radius) {
        distance = radius;
    }

    if (distance_square <= radius * radius) {
        auto dot_prod = delta_x * (vx[i] - vx[j]) + delta_y * (vy[i] - vy[j]);
        auto scalar = 2 / (m[i] + m[j]) * dot_prod / distance_square;
        vx[i] -= scalar * delta_x * m[j];
        vy[i] -= scalar * delta_y * m[j];
        vx[j] += scalar * delta_x * m[i];
        vy[j] += scalar * delta_y * m[i];
        // now relax the distance a bit: after the collision, there must be
        // at least (ratio * radius) between them
        x[i] += delta_x / distance * ratio * radius / 2.0;
        y[i] += delta_y / distance * ratio * radius / 2.0;
        x[j] -= delta_x / distance * ratio * radius / 2.0;
        y[j] -= delta_y / distance * ratio * radius / 2.0;
    } else {
        // update acceleration only when no collision
        auto scalar = gravity / distance_square / distance;
        ax[i] -= scalar * delta_x * m[j];
        ay[i] -= scalar * delta_y * m[j];
        ax[j] += scalar * delta_x * m[i];
        ay[j] += scalar * delta_y * m[i];
    }
}

__global__ void update_for_tick(double elapse, double gravity, double position_range, double radius, int thread_num) {
    int rank = blockIdx.x * blockDim.x + threadIdx.x;
    int task = BODIES / thread_num;
    int start = rank * task;
    int end = start + task;
    if (rank == thread_num - 1) {
        end = BODIES;
    }
    // printf("Rank: %d\n", rank);

    for (int i = start; i < end; ++i) {
        for (int j = i + 1; j < BODIES; ++j) {
            if (j > start && j < start + task) {
                if (j <= i) continue;
            }
            check_and_update(i, j, radius, gravity);
        }
    }

    // Synchronize all the threads in this point
    __syncthreads();

    for (int i = start; i < end; ++i) {
        update_for_tick(elapse, position_range, radius, i);
    }


}

int main(int argc, char **argv) {
    int thread_num;

    if (argc == 1) {
        thread_num = 1;  // sequential by default
    } else if (argc == 2) {
        thread_num = atoi(*(argv + 1));
    } else {
        std::cerr << "usage: " << argv[0] << " <thread number> " << std::endl;
        return 0;
    }

    // Meta data
    static double gravity = 100;
    static double space = 800;
    static double radius = 5;
    // static int bodies = 200;
    static double elapse = 0.01;
    static ImVec4 color = ImVec4(1.0f, 1.0f, 0.4f, 1.0f);
    static double max_mass = 50;

    // Initialize the data --- random
    std::random_device device;
    std::default_random_engine engine{device()};
    std::uniform_real_distribution<double> position_dist{0, space};
    std::uniform_real_distribution<double> mass_dist{0, max_mass};

    for (auto &i : x) {
        i = position_dist(engine);
    }
    for (auto &i : y) {
        i = position_dist(engine);
    }
    for (auto &i : m) {
        i = mass_dist(engine);
    }

    size_t duration = 0;
    int count = 0;  // The number of iteration

    graphic::GraphicContext context{"Assignment 3"};
    context.run([&](graphic::GraphicContext *context [[maybe_unused]], SDL_Window *) {
        auto io = ImGui::GetIO();
        ImGui::SetNextWindowPos(ImVec2(0.0f, 0.0f));
        ImGui::SetNextWindowSize(io.DisplaySize);
        ImGui::Begin("Assignment 3", nullptr,
                     ImGuiWindowFlags_NoMove
                     | ImGuiWindowFlags_NoCollapse
                     | ImGuiWindowFlags_NoTitleBar
                     | ImGuiWindowFlags_NoResize);
        ImDrawList *draw_list = ImGui::GetWindowDrawList();
        ImGui::Text("Application average %.3f ms/frame (%.1f FPS)", 1000.0f / ImGui::GetIO().Framerate,
                    ImGui::GetIO().Framerate);
        // ImGui::DragFloat("Space", &current_space, 10, 200, 1600, "%f");
        // ImGui::DragFloat("Gravity", &gravity, 0.5, 0, 1000, "%f");
        // ImGui::DragFloat("Radius", &radius, 0.5, 2, 20, "%f");
        // ImGui::DragInt("Bodies", &current_bodies, 1, 2, 100, "%d");
        // ImGui::DragFloat("Elapse", &elapse, 0.001, 0.001, 10, "%f");
        // ImGui::DragFloat("Max Mass", &current_max_mass, 0.5, 5, 100, "%f");
        // ImGui::ColorEdit4("Color", &color.x);

        {
            using namespace std::chrono;

            const ImVec2 p = ImGui::GetCursorScreenPos();

            /* Start calculation */
            auto begin = high_resolution_clock::now();           
            // Cuda initialization
            update_for_tick<<<1, thread_num>>>(elapse, gravity, space, radius, thread_num);
            
            // Wait for computation to finish
            hipDeviceSynchronize();

            auto end = high_resolution_clock::now();
            /* Finish calculation */ 

            count++;
            duration += duration_cast<nanoseconds>(end - begin).count();
            if (count == ITERATION) {
                std::cout << ITERATION << " elapse with " << duration << " nanoseconds\n";
                double speed = double(ITERATION) / double(duration) * 1e9;
                std::cout << "speed: " << speed << " iterations per second" << std::endl;
                count = 0;
                duration = 0;
            }      

            for (int i = 0; i < BODIES; ++i) {
                auto px = p.x + x[i];
                auto py = p.y + y[i];
                draw_list->AddCircleFilled(ImVec2(px, py), radius, ImColor{color});
            }
            
            for (int i = 0; i < BODIES; i++) {
                ax[i] = 0;
                ay[i] = 0;
            }
        }
        ImGui::End();
    });
    hipDeviceReset();
    return 0;
}
