#include "hip/hip_runtime.h"
#include <graphic/graphic.hpp>
#include <imgui_impl_sdl.h>
#include <cstring>
#include <chrono>
#include <iostream>
#include <cmath>
#include <hdist/hdist.hpp>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define ITERATION 10

// Parameters
#define ROOMSIZE 300
#define SOURCEX 150
#define SOURCEY 150
#define SOURCETEMP 100
#define BOARDTEMP 36
#define TOLERANCE 0.1

__device__ __managed__ double data0[ROOMSIZE * ROOMSIZE];
__device__ __managed__ double data1[ROOMSIZE * ROOMSIZE];
__device__ __managed__ bool finished = false;
__device__ __managed__ int thread_num = 1;
__device__ __managed__ bool first_buffer = true;

__device__ double update_single(int i, int j) {
    double temp = 0.0;
    if (i == 0 || j == 0 || i == ROOMSIZE - 1 || j == ROOMSIZE - 1) {
        temp = double(BOARDTEMP);
    } else if (i == SOURCEX && j == SOURCEY) {
        temp = double(SOURCETEMP);
    } else {
        bool stable = false;
        double* data = (first_buffer) ? data0 : data1;
        double sum = data[(i + 1) * ROOMSIZE + j] + data[(i - 1) * ROOMSIZE + j] + data[i * ROOMSIZE + (j + 1)] + data[i * ROOMSIZE + (j - 1)];
        temp = 0.25 * sum;
        stable = std::fabs(data[i * ROOMSIZE + j] - temp) < TOLERANCE;
        finished &= stable;
        // if (temp != 0.0) printf("i: %d j: %d, temp: %lf up:%lf down: %lf left:%lf right:%lf \n", i, j, temp, data[(i + 1) * ROOMSIZE + j], data[(i - 1) * ROOMSIZE + j], data[i * ROOMSIZE + (j + 1)], data[i * ROOMSIZE + (j - 1)]);
    }
    return temp;
}

__global__ void calculate() {  
    int rank = blockIdx.x * blockDim.x + threadIdx.x;
    int task = int(ROOMSIZE) / thread_num;
    int remain = int(ROOMSIZE) % thread_num;    
    int start = rank * task + remain;
    int end = start + task;

    double* writeData = (first_buffer) ? data1 : data0; 

    if (rank == 0) {
        start = 0;
    }

    // printf("rank: %d start: %d end: %d task: %d", rank, int(start), int(end), task);

    for (int i = start; i < end; ++i) {
        for (int j = 0; j < ROOMSIZE; ++j) {
            double temp = update_single(i, j);
            writeData[i * ROOMSIZE + j] = temp;
        }
    }
}

__host__ ImColor temp_to_color(double temp) {
    auto value = static_cast<uint8_t>(temp / 100.0 * 255.0);
    return {value, 0, 255 - value};
}

__host__ void init_data() {
    for (int i = 0; i < ROOMSIZE; ++i) {
        for (int j = 0; j < ROOMSIZE; ++j) {
            if (i == 0 || j == 0 || i == ROOMSIZE - 1 || j == ROOMSIZE - 1) {
                data0[i * ROOMSIZE + j] = BOARDTEMP;
            } else if (i == SOURCEX && j == SOURCEY) {
                data0[i * ROOMSIZE + j] = SOURCETEMP;
            } else {
                data0[i * ROOMSIZE + j] = 0;
            }
        }
    }
}

int main(int argc, char **argv) {

    // Initialize the tempature
    init_data();  // strange bug, after init_data, thread_num becomes zero
    
    // Print the buffer data
    // for (int i = 0; i < ROOMSIZE; ++i) {
    //     for (int j = 0; j < ROOMSIZE; ++j) {
    //         printf("%lf ", data0[i * ROOMSIZE + j]);
    //     }
    //     printf("\n");
    // }
    
    if (argc == 1) {
        thread_num = 1;  // sequential by default
    } else if (argc == 2) {
        thread_num = atoi(*(argv + 1));
    } else {
        std::cerr << "usage: " << argv[0] << " <thread number> " << std::endl;
        return 0;
    }

    // Decide the graph size
    static float block_size = 2;
    int duration = 0;
    int count = 0;  // The number of iteration

    bool first = true;
    bool info = false;

    static std::chrono::high_resolution_clock::time_point begin, end;
    graphic::GraphicContext context{"Assignment 4"};
    context.run([&](graphic::GraphicContext *context [[maybe_unused]], SDL_Window *) {
        using namespace std::chrono;
        auto io = ImGui::GetIO();
        ImGui::SetNextWindowPos(ImVec2(0.0f, 0.0f));
        ImGui::SetNextWindowSize(io.DisplaySize);
        ImGui::Begin("Assignment 4", nullptr,
                    ImGuiWindowFlags_NoMove
                    | ImGuiWindowFlags_NoCollapse
                    | ImGuiWindowFlags_NoTitleBar
                    | ImGuiWindowFlags_NoResize);
        ImDrawList *draw_list = ImGui::GetWindowDrawList();
        ImGui::Text("Application average %.3f ms/frame (%.1f FPS)", 1000.0f / ImGui::GetIO().Framerate,
                    ImGui::GetIO().Framerate);
        ImGui::DragFloat("Block Size", &block_size, 0.01, 0.1, 10, "%f");

        if (first) {
            first = false;
            finished = false;
            begin = std::chrono::high_resolution_clock::now();
        }

        if (!finished) {

            /* Start calculation */
            auto beginCal = high_resolution_clock::now();
            finished = true;

            // Computation
            calculate<<<1, thread_num>>>();

            // Wait for computation to finish
            hipDeviceSynchronize();
            
            first_buffer = !first_buffer;
            auto endCal = high_resolution_clock::now();
            /* Finish calculation */
            count++;                
            duration += duration_cast<nanoseconds>(endCal - beginCal).count();
            if (count % ITERATION == 0) {
                std::cout << ITERATION << " elapse with " << duration << " nanoseconds\n";
                double speed = double(ITERATION) / double(duration) * 1e9;
                std::cout << "speed: " << speed << " iterations per second" << std::endl;
                duration = 0;
            }

            if (finished) end = high_resolution_clock::now();
        
        } else {
            long time = duration_cast<nanoseconds>(end - begin).count();
            ImGui::Text("stabilized in %ld ns", time);
            if (!info) {
                std::cout << "stabilized in " << time << " ns and " << count << " iterations" << std::endl;
                info = true;
            }
        }

        const ImVec2 p = ImGui::GetCursorScreenPos();
        double* data = (first_buffer) ? data0 : data1;
        float x = p.x + block_size, y = p.y + block_size;
        for (int i = 0; i < ROOMSIZE; ++i) {
            for (int j = 0; j < ROOMSIZE; ++j) {
                double temp = data[i * ROOMSIZE + j];
                auto color = temp_to_color(temp);
                draw_list->AddRectFilled(ImVec2(x, y), ImVec2(x + block_size, y + block_size), color);
                y += block_size;
            }
            x += block_size;
            y = p.y + block_size;
        }
        ImGui::End();
    });        
    hipDeviceReset();
    return 0;
}
